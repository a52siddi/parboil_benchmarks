#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#ifdef __MCUDA__
#include <mcuda.h>
#endif

#include "matrixMul.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void randomInit(float*, int);

__global__ void matrixMul(float* C, float* A, float* B, int wA, int wB);

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    (hipMalloc((void**) &d_A, mem_size_A));
    float* d_B;
    (hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    (hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice) );
    (hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice) );

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    (hipMalloc((void**) &d_C, mem_size_C));
    
    // setup execution parameters
    dim3 threads;
    threads.x = threads.y = BLOCK_SIZE;
    dim3 grid;
    grid.x = WC / threads.x;
    grid.y = HC / threads.y;
    threads.z = grid.z = 1;

    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);

    // allocate mem for the result on host side
    float* h_C = (float*) malloc(mem_size_C);

    // copy result from device to host
    (hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost) );

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    (hipFree(d_A));
    (hipFree(d_B));
    (hipFree(d_C));
    return 0;
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    int i;
    for (i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

