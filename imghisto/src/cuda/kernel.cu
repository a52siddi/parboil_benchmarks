#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2012-2012 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <kernel.h>

__global__ void histo_R_per_block_kernel(unsigned int* histo,
                                         unsigned int* data,
                                         int size, int BINS, int BINSp, int R);

void
histo_R_per_block(unsigned int* histo, // Output histogram on device
                  unsigned int* data,  // Input data on device
                  int size,            // Input data size
                  int NUM_BLOCKS,      // Number of GPU thread blocks
                  int THREADS,         // Number of GPU threads per block
                  int BINS,   // Number of histogram bins to use
                  int BINSp,  // Number of histogram bins to allocate
                  int R,      // Number of histograms per thread block
                  struct pb_TimerSet *timers)
{
  /* Clear output */
  pb_SwitchToTimer(timers, pb_TimerID_COPY);
  hipMemset(histo, 0, BINS*sizeof(unsigned int));

  /* Launch kernel */
  pb_SwitchToTimer(timers, pb_TimerID_KERNEL);
  int shmem_bytes = (BINSp * R) * sizeof(int);
  histo_R_per_block_kernel<<<dim3(NUM_BLOCKS), dim3(THREADS), shmem_bytes>>>
    (histo, data, size, BINS, BINSp, R);

  /* Check for errors */
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    exit(-1);
  }

  /* Synchronize so that subsequent timer measurements are accurate */
  hipDeviceSynchronize();
}

// Constants
#define WARP_SIZE 32

// Dynamic shared memory allocation
extern __shared__ unsigned int Hs[];

///////////////////////////////////////////////////////////////////////////////
// R-per-block approach
// Replication + Padding + Interleaved read access
//
// histo:	Final histogram in global memory
// data:	Input image. Pixels are stored in 4-byte unsigned int
// size:	Input image size (number of pixels)
// BINS:	Number of histogram bins
// BINSp:	Histogram size including padding
// R:		Replication factor
//
// This function was developed at the University of Córdoba and
// contributed by Juan Gómez-Luna.
///////////////////////////////////////////////////////////////////////////////
__global__ void histo_R_per_block_kernel(unsigned int* histo,
                                         unsigned int* data,
                                         int size, int BINS, int BINSp, int R)
{
  // Block and thread index
  const int bx = blockIdx.x;
  const int tx = threadIdx.x;
  // Warp and lane
  const unsigned int warpid = tx >> 5;
  const unsigned int lane = tx & 31;	

  // Offset to per-block sub-histograms
  const unsigned int off_rep = BINSp * (tx % R);

  // Constants for interleaved read access
  const int warps_block = blockDim.x / WARP_SIZE;
  const int begin = (size / warps_block) * warpid + WARP_SIZE * bx + lane;
  const int end = (size / warps_block) * (warpid + 1);
  const int step = WARP_SIZE * gridDim.x;
  // Constants for naive read access
  /*const int begin = bx * blockDim.x + tx;
    const int end = size;
    const int step = blockDim.x * gridDim.x;*/

  // Sub-histograms initialization
  for(int pos = tx; pos < BINSp*R; pos += blockDim.x) Hs[pos] = 0;

  __syncthreads();	// Intra-block synchronization

  // Main loop
  for(int i = begin; i < end; i += step){
    // Global memory read
    unsigned int d = data[i];

    // Atomic vote in shared memory
    atomicAdd(&Hs[off_rep + ((d * BINS) >> 12)], 1);

  }

  __syncthreads();	// Intra-block synchronization

  // Merge per-block histograms and write to global memory
  for(int pos = tx; pos < BINS; pos += blockDim.x){
    unsigned int sum = 0;
    for(int base = 0; base < BINSp*R; base += BINSp)
      sum += Hs[base + pos];
    // Atomic addition in global memory
    atomicAdd(histo + pos, sum);
  }
}


